#include <stdio.h>
#include <hip/hip_runtime.h>
//Kernel to initialize matrix and compute number of even values in the matrix
__global__ void dkernel(unsigned *matrix, int *evenNum) {
    //initializing the matrix 
    //matrix spots are filled with their respective id's 
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    matrix[id] = id;
    //if the matrix value has no remainder when divided by 2 it is even
    //atomicAdd guarentees add operation is performed without interference from other threads. 
    //No other thread can access this address until the operation is complete
    if((matrix[id] % 2) == 0) {
        atomicAdd(evenNum, 1);
    }
}
//kernel to compute the square of a given matrix
__global__ void square ( unsigned *matrix, unsigned *result, unsigned matrixsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned jj = 0; jj < matrixsize; ++jj) 
    {
        for (unsigned kk = 0; kk < matrixsize; ++kk) 
        {
            result[id * matrixsize + jj] += matrix[id * matrixsize + kk] *
            matrix[kk * matrixsize + jj];
        }
    }
}
#define N 8
#define M 8
int main() {
    //*****EXERCISE 1*****
    dim3 block(N, M, 1);
    unsigned *matrix, *hmatrix, *resultGPU, *sqauredResult;
    int *evenNum;
    int count;
    //memory is allocated
    hipMalloc(&matrix, N * M * sizeof(unsigned));
    hipMalloc(&evenNum, sizeof(int));
    hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));
    //kernel called
    dkernel<<<1, block>>>(matrix, evenNum);
    //transfer between host memory and device memory
    hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipMemcpy(&count, evenNum, sizeof(int), hipMemcpyDeviceToHost);
    //matrix printed
    printf("Original Matrix:\n");
    for (unsigned ii = 0; ii < N; ++ii) 
    {
        for (unsigned jj = 0; jj < M; ++jj) 
        {
            printf("%2d ", hmatrix[ii * M + jj]);
        }
        printf("\n");
    }
    //number of even value in matrix printed
    printf("Number of even values in the NxN matrix: %d\n\n", count);
    

    //*****EXERCISE 2*****
    //memory is allocated
    sqauredResult = (unsigned *)malloc(N * M * sizeof(unsigned));
    hipMalloc(&resultGPU, N * M * sizeof(unsigned));
    //kernel called
    square<<<1, N>>>(matrix, resultGPU, N);
    //trasnfer between host memory and device memory
    hipMemcpy(sqauredResult, resultGPU, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
    //squared matrix printed
    printf("Sqaure Matrix:\n");
    for (unsigned ii = 0; ii < N; ++ii) 
    {
        for (unsigned jj = 0; jj < N; ++jj) 
        { 
            printf("%2d ", sqauredResult[ii * N + jj]); 
        } 
        printf("\n"); 
    } 

    return 0;
}